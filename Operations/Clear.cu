#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <string>

#include <csignal> 
#include "Clear.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void clear_layer_kernel(double *aA, double *bB, bool interconnection, int clrLayer, int length, int *cC)
{
    int node = blockIdx.x*blockDim.x + threadIdx.x;
    int IDX;

    if (interconnection)
    {
        int connection = blockIdx.y*blockDim.y + threadIdx.y;
        IDX = internodal_data_list_indexer(node, clrLayer, connection, cC);
    }
    else
    {
        IDX = nodal_data_list_indexer(node, clrLayer, cC);
    };

    if (IDX < length)
    {
        bB[IDX] = 0.0;
    };
}

namespace Clear
{
    vector<double> clear_layer(vector<double> A, int _layer, vector<int> architecture, int numb_layers, bool interconnection, bool forward)
    {
        double *cudaA;
        double *cudaB;
        int *cudaC;

        int layer = (numb_layers-1) - _layer;

        vector<double> vecA = A;

        int vecLength = vecA.size();

        vector<double> vecB = A;

        // Allocate memory to gpu.
        if (hipMalloc((void**)&cudaA, vecLength*sizeof(double))!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            return {};
        };
        if (hipMalloc((void**)&cudaB, vecLength*sizeof(double))!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            hipFree(cudaA);
            return {};
        };
        if (hipMalloc((void**)&cudaC, numb_layers*sizeof(int))!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            hipFree(cudaA);
            hipFree(cudaB);
            return {};
        };

        // Supply allocated memory with data from host.
        if (hipMemcpy(cudaA, vecA.data(), vecLength*sizeof(double), hipMemcpyHostToDevice)!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            hipFree(cudaA);
            hipFree(cudaB);
            hipFree(cudaC);
            return {};
        };
        if (hipMemcpy(cudaB, vecB.data(), vecLength*sizeof(double), hipMemcpyHostToDevice)!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            hipFree(cudaA);
            hipFree(cudaB);
            hipFree(cudaC);
            return {};
        };
        if (hipMemcpy(cudaC, architecture.data(), numb_layers*sizeof(int), hipMemcpyHostToDevice)!=hipSuccess)
        {
            std::cout<<"Allocated memory did not accept given data!\n";
            hipFree(cudaA);
            hipFree(cudaB);
            hipFree(cudaC);
            return {};
        };

        // Specify number of threads to run on GPU.
        int threadsPerBlock = 16;
        int blocksPerGrid1;
        int blocksPerGrid2;

        int length_of_vec = vecLength;
        int clrLayer;

        if (forward)
        {
            blocksPerGrid1 = architecture[_layer] / threadsPerBlock + 1;
            clrLayer = _layer + 1;
            if (interconnection)
            {
                blocksPerGrid2 = architecture[_layer+1] / threadsPerBlock + 1;
            }
            else
            {
                blocksPerGrid2 = 1;
            };
        }
        else
        {
            blocksPerGrid1 = architecture[layer] / threadsPerBlock + 1;
            clrLayer = layer - 1;
            if (interconnection)
            {
                blocksPerGrid2 = architecture[layer-1] / threadsPerBlock + 1;
            }
            else
            {
                blocksPerGrid2 = 1;
            };
        };

        dim3 blockDIM(threadsPerBlock, threadsPerBlock);
        dim3 gridDIM(blocksPerGrid1, blocksPerGrid2);

        // Run calculation on device kernel.
        clear_layer_kernel<<<gridDIM,blockDIM>>>(cudaA,cudaB,interconnection,clrLayer,length_of_vec,cudaC);

        // Catch possible gpu errors.
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        // Copy vectors from GPU to host.
        if (hipMemcpy(vecB.data(), cudaB, vecLength*sizeof(double), hipMemcpyDeviceToHost) != hipSuccess)
        {
	        cout<<"Alloacted memory did not accept supplied data!\n";
	        hipFree(cudaA);
	        hipFree(cudaB);
            hipFree(cudaC);
	        return {};
        }

        // Unallocate all pointers.
        hipFree(cudaA);
        hipFree(cudaB);
        hipFree(cudaC);

        return vecB;
    }
}
